#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <chrono>
#include <fstream>
#include <assert.h>
using namespace std;

#define TIME_NOW std::chrono::high_resolution_clock::now()
#define TIME_DIFF(gran, start, end) std::chrono::duration_cast<gran>(end - start).count()

#include "gpu_thread.h"
#include "cpu_thread.h"

// Used to cross-check answer. DO NOT MODIFY!

int main(int argc, char *argv[])
{
    // Input size of square matrices// Input size of square matrices
    int N;
    string file_name; 
    if (argc < 2) 
        file_name = "data/input_128.in"; 
    else 
        file_name = argv[1]; 
    ifstream input_file; 
    input_file.open(file_name); 
    input_file >> N;
    cout << "Input matrix of size " << N << "\n";
    
    // Input matrix A
    int *matA = new int[N * N];
    for(int i = 0; i < N; ++i)
        for(int j = 0; j < N; ++j)
            input_file >> matA[i * N + j];

    // Input matrix B
    int *matB = new int[N * N];
    for(int i = 0; i < N; ++i)
        for(int j = 0; j < N; ++j)
            input_file >> matB[i * N + j];
    
    auto begin = TIME_NOW;
    // Execute reference program
    int *output_reference = new int[N*(N>>1)];
    multiThread(N, matA, matB, output_reference);
    auto end = TIME_NOW;
    cout << "Reference execution time: " << 
        (double)TIME_DIFF(std::chrono::microseconds, begin, end) / 1000.0 << " ms\n";    

    // Execute gpu version
    int *output_gpu = new int[N*(N>>1)];
    begin = TIME_NOW;
    gpuThread(N, matA, matB, output_gpu);
    end = TIME_NOW;
    cout << "GPU execution time: " << 
        (double)TIME_DIFF(std::chrono::microseconds, begin, end) / 1000.0 << " ms\n";

    for(int i = 0; i < N*(N>>1); ++i)
        if(output_gpu[i] != output_reference[i]) {
            cout << "Mismatch at " << i << "\n";
            cout << "GPU output: " << output_gpu[i] << ", required output: " << output_reference[i] << "\n";
            exit(0);
        }
    input_file.close(); 
}
